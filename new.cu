#include "hip/hip_runtime.h"
# include<stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<Windows.h>


__global__ void addfromGPU(float *A, float *B, float *C, const int n) {
	const int bid = blockIdx.x;
	const int tid = threadIdx.x;
	const int id = tid + bid * blockDim.x;
	//if (id >= 5) return;
	C[id] = A[id] + B[id];
	return;
}

void setGPU() {
	int iDevice = 0;
	hipSetDevice(iDevice);
}

void initiaData(float *addr, int element) {
	for (int i = 0; i < element; i ++) {
		addr[i] = (float)(rand() & 0xFFF) / 10.f;
	}
	return;
}

int main(){
	//���û��豸
	setGPU();

	//���������ڴ���豸�ڴ棬����ʼ��
	int iElement = 5120;  //set the size of element
	size_t bytesize = iElement * sizeof(float);  //�����ڴ�ռ�

	//���������ڴ�ռ�
	float *fpHost_A, *fpHost_B, *fpHost_C;
	fpHost_A = (float*)malloc(bytesize);
	fpHost_B = (float*)malloc(bytesize);
	fpHost_C = (float*)malloc(bytesize);
	//��ʼ�������ڴ�ռ�
	memset(fpHost_A, 0, bytesize);
	memset(fpHost_B, 0, bytesize);
	memset(fpHost_C, 0, bytesize);

	//�����豸�ڴ�ռ�
	float *fpDevice_A, *fpDevice_B, *fpDevice_C;
	hipMalloc((float**)&fpDevice_A,bytesize);
	hipMalloc((float**)&fpDevice_B, bytesize);
	hipMalloc((float**)&fpDevice_C, bytesize);
	//��ʼ���豸�ڴ�ռ�
	hipMemset(fpDevice_A, 0, bytesize);
	hipMemset(fpDevice_B, 0, bytesize);
	hipMemset(fpDevice_C, 0, bytesize);

	//��ʼ�������ڴ�����
	srand(666);
	initiaData(fpHost_A, iElement);
	initiaData(fpHost_B, iElement);

	//���������Ƶ��豸
	hipMemcpy(fpDevice_A, fpHost_A, bytesize, hipMemcpyHostToDevice);
	hipMemcpy(fpDevice_B, fpHost_B, bytesize, hipMemcpyHostToDevice);

	//���ú˺������豸�н��м���
	dim3 block(512);
	dim3 grid(iElement / 512);

	//count time
	double start = clock();
	addfromGPU <<<grid, block>>> (fpDevice_A, fpDevice_B, fpDevice_C, iElement);
	double end = clock();
	printf("%lf\n\n", end-start);

	//������ȥ��չʾ
	hipMemcpy(fpHost_C, fpDevice_C, bytesize, hipMemcpyDeviceToHost);;
	for (int i = 0; i < 10; i++)
	{
		printf("%lf, %lf, %lf\n", fpHost_A[i], fpHost_B[i], fpHost_C[i]);
	}

	free(fpHost_A);
	free(fpHost_B);
	free(fpHost_C);
	hipFree(fpDevice_A);
	hipFree(fpDevice_B);
	hipFree(fpDevice_C);
	return 0;
}